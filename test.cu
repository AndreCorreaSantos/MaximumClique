
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10000000
// block --> 256 threads 
// CUDA kernel to add elements of two arrays
__global__ void vector_add(float *out, const float *a, const float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        out[index] = a[index] + b[index];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f; // Example values
        b[i] = 2.0f; // Example values
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Execute the kernel
    int blockSize = 256; // Number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // Number of blocks
    vector_add<<<numBlocks, blockSize>>>(d_out, d_a, d_b, N);

    // Copy the results back to the host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Print a few results
    for (int i = 0; i < 100; i++) {
        printf("%f ", out[i]);
    }
    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

    return 0;
}
