#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <limits>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;


class Route{
    public:
        __host__ __device__ Route() {}
        __host__ __device__ int size;
        __host__ __device__ int *stops;
        __host__ __device__ int totalCost;
        __host__ __device__ bool valid;
}



void read_demands(ifstream &file, int *demands, int num_vertices) {
    for (int i = 1; i < num_vertices; i++) {
        int stop, demand;
        file >> stop >> demand;
        demands[stop] = demand;
        cout << "stop, demand: " << stop << ", " << demand << "\n";
    }
}

void read_routes(ifstream &file, int num_vertices, int *route_matrix) {
    int num_routes;
    file >> num_routes;

    for (int i = 0; i < num_vertices; i++) {
        for (int j = 0; j < num_vertices; j++) {
            route_matrix[i * num_vertices + j] = 0; // initializing all costs as 0
        }
    }

    for (int i = 0; i < num_routes; i++) {
        int u, v, cost;
        file >> u >> v >> cost;
        route_matrix[u * num_vertices + v] = cost;
        route_matrix[v * num_vertices + u] = cost;
    }
}

void write_route_matrix(int *routes, int num_vertices, const string &filename) {
    ofstream outFile(filename);
    if (!outFile.is_open()) {
        cout << "Error opening output file." << endl;
        return;
    }

    for (int i = 0; i < num_vertices; i++) {
        for (int j = 0; j < num_vertices; j++) {
            outFile << routes[i * num_vertices + j] << " ";
        }
        outFile << "\n";
    }
    outFile.close();
}

void permute(vector<int> &route, int start, vector<vector<int>> &permutations) {
    if (start == route.size()) {
        vector<int> full_route;
        full_route.push_back(0);
        full_route.insert(full_route.end(), route.begin(), route.end());
        full_route.push_back(0);
        permutations.push_back(full_route);
    } else {
        for (int i = start; i < route.size(); i++) {
            swap(route[start], route[i]);
            permute(route, start + 1, permutations);
            swap(route[start], route[i]); // backtrack
        }
    }
}

vector<vector<int>> generate_permutations(int num_points) {
    vector<vector<int>> permutations;
    vector<int> route(num_points - 1); // Only permute num_points-1 elements

    for (int i = 1; i < num_points; i++) {
        route[i - 1] = i;
    }

    permute(route, 0, permutations);
    return permutations;
}

__global__ void filter_routes(int *permutations, int *route_matrix, int *demands, int *valid_routes, int max_weight, int max_stops, int max_route_size,int n_permutations) {
    int tId = (threadIdx.x + blockIdx.x * blockDim.x);


    if(tId>n_permutations-1){
        return;
    }
    int route_index = (tId*max_route_size) + 1;


    int cost = 0;
    int weight = 0;
    for(int i = 0; i<max_route_size-1; i++){

        int current_stop = permutations[route_index+i];    
        int next_stop = permutations[route_index+i+1];

    }
}

cpu_Route get_cheapest(vector<cpu_Route> valid_routes, vector<vector<int>> route_matrix) {
    int lowest_cost = numeric_limits<int>::max();
    cpu_Route cheapest_route;
    for (int i = 0; i < int(valid_routes.size()); i++) {
        if (valid_routes[i].totalCost < lowest_cost) {
            lowest_cost = valid_routes[i].totalCost;
            cheapest_route = valid_routes[i];
        }
    }
    return cheapest_route;
}

void debug_route(vector<int> route) {
    for (int i = 0; i < int(route.size()); i++) {
        cout << route[i] << "\n";
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <filename>\n";
        return 1;
    }
    string file_name = argv[1];
    ifstream file(file_name);

    int num_vertices;
    file >> num_vertices;
    int *demands = (int *)malloc(num_vertices * sizeof(int));
    read_demands(file, demands, num_vertices);

    int *route_matrix = (int *)malloc(num_vertices * num_vertices * sizeof(int));
    read_routes(file, num_vertices, route_matrix);

    vector<vector<int>> permutations = generate_permutations(num_vertices);
    int num_routes = permutations.size();

    size_t total_size = 0;
    for (const auto &vec : permutations) {
        total_size += vec.size();
    }

    int *flat_permutations = new int[total_size];
    size_t index = 0;
    for (const auto &vec : permutations) {
        for (int val : vec) {
            flat_permutations[index++] = val;
        }
    }

    int max_weight = 15;
    int max_stops = 7;
    int *valid_routes = new int[max_stops * num_routes];

    int *d_valid_routes, *d_permutations, *d_route_matrix, *d_demands;

    int max_route_size = 2 * num_vertices + 1;
    hipMalloc((void **)&d_valid_routes, sizeof(int) * max_route_size * num_routes);
    hipMalloc((void **)&d_permutations, sizeof(int) * total_size);
    hipMalloc((void **)&d_route_matrix, sizeof(int) * num_vertices * num_vertices);
    hipMalloc((void **)&d_demands, sizeof(int) * num_vertices);

    hipMemcpy(d_permutations, flat_permutations, sizeof(int) * total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_route_matrix, route_matrix, sizeof(int) * num_vertices * num_vertices, hipMemcpyHostToDevice);
    hipMemcpy(d_demands, demands, sizeof(int) * num_vertices, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_routes + blockSize - 1) / blockSize;

    filter_routes<<<numBlocks, blockSize>>>(d_permutations, d_route_matrix, d_demands, d_valid_routes, max_weight, max_stops, max_route_size);
    hipMemcpy(valid_routes, d_valid_routes, sizeof(int) * max_route_size * num_routes, hipMemcpyDeviceToHost);

    file.close();
    return 0;
}
