
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){ // isso é um kernel
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();  // número de threads em x e y que vão executar o kernel
    return 0;
}