#include <fstream>
#include <iostream>
#include <limits>
#include <vector>
#include <map>
#include <hip/hip_runtime.h>

using namespace std;

class Route {
public:
    __host__ __device__ Route() : size(0), stops(nullptr), totalCost(0), valid(false) {}

    __host__ __device__ Route(int size, int* stops) : size(size), stops(stops), totalCost(0), valid(false) {}

    __host__ Route(int max_stops) : size(0), totalCost(0), valid(false) {
        hipMallocManaged(&stops, max_stops * sizeof(int));
        hipMemset(stops, 0, max_stops * sizeof(int));
    }

    __host__ __device__ ~Route() {
        if (stops != nullptr) {
            hipFree(stops);
        }
    }

    int size;
    int* stops;
    int totalCost;
    bool valid;
};


void read_demands(ifstream& file, int* demands, int num_vertices) {
    for (int i = 1; i < num_vertices; i++) {
        int stop, demand;
        file >> stop >> demand;
        demands[stop] = demand;
        cout << "stop, demand: " << stop << ", " << demand << "\n";
    }
}

void read_routes(ifstream& file, int num_vertices, int* route_matrix) {
    int num_routes;
    file >> num_routes;

    for (int i = 0; i < num_vertices; i++) {
        for (int j = 0; j < num_vertices; j++) {
            route_matrix[i * num_vertices + j] = 0; // initializing all costs as 0
        }
    }

    for (int i = 0; i < num_routes; i++) {
        int u, v, cost;
        file >> u >> v >> cost;
        route_matrix[u * num_vertices + v] = cost;
        route_matrix[v * num_vertices + u] = cost;
    }
}

void write_routes(const vector<Route>& routes, const string& filename) {
    ofstream outFile(filename);

    if (!outFile.is_open()) {
        cout << "Error opening output file." << endl;
        return;
    }

    for (const Route& route : routes) {
        for (int j = 0; j < route.size; j++) {
            outFile << route.stops[j] << " ";
        }
        outFile << " | " << route.totalCost;
        outFile << endl;
    }

    outFile.close();
}

void permute(vector<int>& route, int start, vector<vector<int>>& permutations) {
    if (start == route.size()) {
        vector<int> full_route;
        full_route.push_back(0);
        full_route.insert(full_route.end(), route.begin(), route.end());
        full_route.push_back(0);
        permutations.push_back(full_route);
    } else {
        for (int i = start; i < route.size(); i++) {
            swap(route[start], route[i]);
            permute(route, start + 1, permutations);
            swap(route[start], route[i]); // backtrack
        }
    }
}

vector<vector<int>> generate_permutations(int num_points) {
    vector<vector<int>> permutations;
    vector<int> route(num_points - 1); // Only permute num_points-1 elements

    for (int i = 1; i < num_points; i++) {
        route[i - 1] = i;
    }

    permute(route, 0, permutations);
    return permutations;
}

__device__ void insert_into_route(Route& route, int capacity, int index, int value) {
    if (route.size >= capacity) {
        return;
    }

    // Shift elements to the right to make space for the new element
    for (int i = route.size; i > index; --i) {
        route.stops[i] = route.stops[i - 1];
    }

    route.stops[index] = value;

    route.size++;
}

__global__ void filter_routes_kernel(Route* d_routes, int* route_matrix, int* demands, Route* valid_routes, int max_weight, int num_permutations, int num_vertices, int max_route_size) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId > num_permutations-1) return;

    Route& route = d_routes[tId];
    int cost = 0;
    int weight = 0;
    bool valid = true;

    for (int i = 0; i < route.size - 1; i++) {
        int current_stop = route.stops[i];
        int next_stop = route.stops[i + 1];

        int current_cost = route_matrix[current_stop * num_vertices + next_stop];
        int next_weight = demands[next_stop];
        bool over_weight = weight + next_weight > max_weight;

        if (current_cost == 0 || over_weight) {
            // Check return path to the depot
            int return_cost = route_matrix[current_stop * num_vertices];
            int to_next_cost = route_matrix[next_stop];

            if (return_cost == 0 || to_next_cost == 0) {
                valid = false;
                break;
            }

            // Insert zero after the current stop if not back-to-back zeros
            insert_into_route(route, max_route_size, i + 1, 0);

            weight = 0; // resetting the weight as the vehicle returned to the origin
            next_weight = 0;
            current_cost = return_cost;
        }

        weight += next_weight;
        cost += current_cost;
    }

    if (valid) {
        route.totalCost = cost;
        route.valid = true;
        valid_routes[tId] = route;  // Store the valid route
    }
}

Route get_cheapest(const vector<Route>& valid_routes) {
    int lowest_cost = numeric_limits<int>::max();
    Route cheapest_route;

    for (const auto& route : valid_routes) {
        if (route.valid && route.totalCost < lowest_cost) {
            lowest_cost = route.totalCost;
            cheapest_route = route;
        }
    }

    return cheapest_route;
}

void debug_route(const vector<int>& route) {
    for (int stop : route) {
        cout << stop << " ";
    }
    cout << endl;
}

void debug_route_matrix(int* route_matrix, int num_vertices) {
    cout << "Route Matrix:" << endl;
    for (int i = 0; i < num_vertices; i++) {
        for (int j = 0; j < num_vertices; j++) {
            cout << route_matrix[i * num_vertices + j] << " ";
        }
        cout << endl;
    }
}

void debug_demands(int* demands, int num_vertices) {
    cout << "Demands:" << endl;
    for (int i = 0; i < num_vertices; i++) {
        cout << "Stop " << i << ": " << demands[i] << endl;
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <filename>\n";
        return 1;
    }

    string file_name = argv[1];
    ifstream file(file_name);

    int num_vertices;
    file >> num_vertices;
    int* demands;
    hipMallocManaged(&demands, num_vertices * sizeof(int));
    read_demands(file, demands, num_vertices);

    debug_demands(demands, num_vertices); // Debugging demands

    int* route_matrix;
    hipMallocManaged(&route_matrix, num_vertices * num_vertices * sizeof(int));
    read_routes(file, num_vertices, route_matrix);

    debug_route_matrix(route_matrix, num_vertices); // Debugging route matrix

    vector<vector<int>> permutations = generate_permutations(num_vertices);

    int num_routes = permutations.size();
    int max_route_size = 15; // setting max_route_size to 15 to test

    Route* host_routes;
    hipMallocManaged(&host_routes, num_routes * sizeof(Route));

    for (int i = 0; i < num_routes; i++) {
        host_routes[i] = Route(max_route_size);
        for (size_t j = 0; j < permutations[i].size(); j++) {
            host_routes[i].stops[j] = permutations[i][j];
        }
        host_routes[i].size = permutations[i].size();
    }

    Route* d_valid_routes;
    hipMallocManaged(&d_valid_routes, num_routes * sizeof(Route));
    
    int count = 0;
    for (int i = 0; i<num_routes; i++){
        for (int j = 0; j<max_route_size-1; j++){
            count += d_valid_routes[i].stops[j];
        }
    }
    cout << count;
    // int blockSize = 256;
    // int numBlocks = (num_routes + blockSize - 1) / blockSize;

    // int max_weight = 15;
    // filter_routes_kernel<<<numBlocks, blockSize>>>(host_routes, route_matrix, demands, d_valid_routes, max_weight, num_routes, num_vertices, max_route_size);
    // cudaDeviceSynchronize();

    // vector<Route> valid_routes(num_routes);
    // cudaMemcpy(valid_routes.data(), d_valid_routes, num_routes * sizeof(Route), cudaMemcpyDeviceToHost);

    // // Write all valid routes to file
    // vector<Route> results;
    // for (const auto& route : valid_routes) {
    //     if (route.valid) {
    //         results.push_back(route);
    //     }
    // }
    // write_routes(results, "debug_cuda.txt");

    // file.close();
    // cudaFree(demands);
    // cudaFree(route_matrix);
    // cudaFree(host_routes);
    // cudaFree(d_valid_routes);

    // return 0;
}
